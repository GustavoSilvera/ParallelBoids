#include "hip/hip_runtime.h"
#include "Flock.hpp"  // Flocks
#include "Tracer.hpp" // Tracer
#include "Utils.hpp"  // Params
#include "Vec.hpp"    // Vec3D
#include <chrono>     // timing threads
#include <omp.h>      // OpenMP
#include <string>     // cout
#include <vector>     // std::vector

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

struct GlobalConstants {
    size_t numBoids;

    float* position;
    float* velocity;
    int* flockID; 
    int* flockSize;
};

__constant__ GlobalConstants cuConstBoidData;
__constant__ ParamsStruct cuConstGlobalParams;

/** @return the result of adding the inputs together */
__device__ float2 operator+(const float2 A, const float2 B)
{ 
    return make_float2(A.x + B.x, A.y + B.y);
}

/** @return the result of subtracting B from A */
__device__ float2 operator-(const float2 A, const float2 B)
{
    return make_float2(A.x - B.x, A.y - B.y);
}

/** @return the result of multiplying each element of A by v */
__device__ float2 operator*(const float2 A, const int v) 
{
    return make_float2(A.x * v, A.y * v);
}

/** @return the result of dividing each element of A by v */
__device__ float2 operator/(const float2 A, const int v) 
{
    return make_float2(A.x / v, A.y / v);
}

/** @return the square of the l2 norm of the specified vector */
__device__ int sizeSqrd (float2 A) 
{
    return A.x*A.x + A.y*A.y;
}

/** @return whether or not A lies beyond a distance of radius R from B */
__device__ bool distGT (const float2 A, const float2 B, const int R)
{
    return sizeSqrd(A-B) > R*R;
}

/** @return whether or not A lies within a distance of radius R from B */
__device__ bool distLT (const float2 A, const float2 B, const int R)
{
    return sizeSqrd(A-B) < R*R;
}

__device__ float2 normalize(const float2 A) {
    return A / (sqrt(double(sizeSqrd(A))));
}

__device__ float2 limitMagnitudeKernel(const float2 A, const double maxMag)
{
    if (sizeSqrd(A) > maxMag * maxMag)
    {
        return normalize(A) * maxMag;
    }
    return A;
}
/** 
 * Computes the acceleration for the boid indexed by index in the current
 * tick.
 */ 
__device__ void 
senseAndPlanKernel(float2 &a1, float2 &a2, float2 &a3, int index) 
{
    int N = cuConstBoidData.numBoids;
    float2* position = (float2*)cuConstBoidData.position;
    float2* velocity = (float2*)cuConstBoidData.velocity;

    BoidParamsStruct boidParams = cuConstGlobalParams.BoidParams;

    float2 posUs = position[index];
    float2 velUs = velocity[index];

    float2 relCOM = make_float2(0.0,0.0);  
    float2 relCOV = make_float2(0.0,0.0);
    float2 sep = make_float2(0.0,0.0);

    int numCloseBy = 0;
    for(size_t i = 0; i < N; i++) {
        float2 posThem = position[i];
        float2 velThem = velocity[i];

        if (i != index 
            && !distGT(posUs,posThem, boidParams.NeighbourhoodRadius)) 
        {
            relCOM = relCOM + posThem;  
            relCOV = relCOV + velThem;
            if (distLT(posUs,posThem,boidParams.CollisionRadius))
            {
                sep = sep - (posThem - posUs);
            }
        }
        numCloseBy++;
    }

    if (numCloseBy > 0)
    {
        a1 = ((relCOM / numCloseBy) - posUs) * (boidParams.Cohesion);
        a2 = sep * (boidParams.Separation); // dosent depent on NumCloseby but makes sense
        a3 = ((relCOV / numCloseBy) - velUs) * (boidParams.Alignment);
    }
}

__device__ void 
actKernel(float2 a1, float2 a2, float2 a3, double deltaTime, int i)
{

    /// NOTE: This function is meant to be independent from all other boids
    /// and thus can be run asynchronously, however it needs a barrier between itself
    /// and the senseAndPlan() device function
    float2* posPtr = &(((float2*)(cuConstBoidData.position))[i]);
    float2* velPtr = &(((float2*)(cuConstBoidData.velocity))[i]);
    BoidParamsStruct params = cuConstGlobalParams.BoidParams;

    float2 acceleration = a1 + a2 + a3; // + a4
    *velPtr = limitMagnitudeKernel((*velPtr + acceleration), params.MaxVel);
    *posPtr = (*posPtr) + ((*velPtr) * deltaTime);
}

__global__ void sensePlanActKernel()
{
    const int index = blockIdx.x + threadIdx.x;
    float2 a1,a2,a3;
    senseAndPlanKernel(a1,a2,a3,index);
    __syncthreads();
    actKernel(a1,a2,a3,cuConstGlobalParams.SimulatorParams.DeltaTime,index);
}


class Simulator
{
  public:
    float* cudaDevicePosition;
    float* cudaDeviceVelocity;
    int* cudaDeviceFlockID;
    int* cudaDeviceFlockSize;

    int numBoids;
    float* position;
    float* velocity;
    int* flockID;
    int* flockSize;
    Simulator()
    {
        Params = GlobalParams.SimulatorParams;
        // Print out status
        std::cout << "Running on " << Params.NumBoids << " boids for " << Params.NumIterations << " iterations in a ("
                  << GlobalParams.ImageParams.WindowX << ", " << GlobalParams.ImageParams.WindowY << ") world with "
                  << Params.NumThreads << " threads" << std::endl;

        // Initialize neighbourhood layout for flocks before use
        Flock::InitNeighbourhoodLayout();
        // Spawn flocks
        for (size_t i = 0; i < Params.NumBoids; i++)
        {
            AllFlocks.push_back(Flock(i, 1));
        }

        // begin tracking which flocks communicate with which
        Tracer::InitFlockMatrix(AllFlocks.size());

        // Allocate and initialize device memory data
        Setup();

        // initialize image frame
        if (Params.RenderingMovie)
        {
            // only allocate memory if we're gonna use it
            I.Init();
        }
    }
    static SimulatorParamsStruct Params;
    std::vector<Flock> AllFlocks;
    std::vector<int> FlockSizes;
    Image I;

    void Simulate()
    {
        double ElapsedTime = 0;
        for (size_t i = 0; i < Params.NumIterations; i++)
        {
            ElapsedTime += Tick();
            std::cout << "Tick: " << i << "\r" << std::flush; // carriage return, no newline
        }
        std::cout << "Finished simulation! Took " << ElapsedTime << "s" << std::endl;
    }

    /**
     * Inserts the value v in the float2 array represented by arr
     * @pre i < |arr|/2
     */
    void insertFloat2(float* arr, Vec2D v, size_t i) {
        arr[2*i] = v[0];
        arr[2*i+1] = v[1];
    }

    /** @return a list of all the boids in the simulation */
    std::vector<Boid> GetAllBoids ()
    {
        return *(AllFlocks.begin()->Neighbourhood.GetAllBoidsPtr());
    }

    void InitBoidDataArrays() {
        std::vector<Boid> AllBoids = GetAllBoids();
        size_t vecSize = sizeof(float) * 2 * numBoids;
        size_t intSize = sizeof(int) * numBoids;

        position = (float*)malloc(vecSize);
        velocity = (float*)malloc(vecSize);
        flockID = (int*)malloc(intSize);
        flockSize = (int*)malloc(intSize);

        for (size_t i=0; i < AllBoids.size(); i++)
        {
            Boid boid = AllBoids[i];
            Vec2D pos = boid.Position;
            Vec2D vel = boid.Velocity;
            int fID = boid.FlockID;
            int fSize = 1;

            insertFloat2(position, pos, i);
            insertFloat2(velocity, vel, i);
            flockID[i] = fID;
            flockSize[i] = fSize;
        }
    }

    /**
     * @pre All Boid arrays have been initialized and are up to date.
     */
    void UpdateBoidPosAndVel() {
        std::vector<Boid> AllBoids = GetAllBoids();
        for (size_t i = 0; i < AllBoids.size(); i++)
        {
            Boid B = AllBoids[i];
            B.Position = position[i];
            B.Velocity = velocity[i];
        }
    }

    /** 
     * Sets up the memory required for the cuda device code
     *
     * @return The time it took to setup. 
     */
    double Setup() {
        auto StartTime = std::chrono::system_clock::now();

        // allocate boid and flock memory
        std::vector<Boid> AllBoids = GetAllBoids();
        numBoids = AllBoids.size();
        size_t vecSize = sizeof(float) * 2 * numBoids;
        size_t intSize = sizeof(int) * numBoids;
        hipMalloc(&cudaDevicePosition, vecSize);
        hipMalloc(&cudaDeviceVelocity, vecSize);
        hipMalloc(&cudaDeviceFlockID, intSize);
        hipMalloc(&flockSize, intSize);

        InitBoidDataArrays();
        
        hipMemcpy(cudaDevicePosition,position,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceVelocity,velocity,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceFlockID,flockID,intSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceFlockSize,flockSize,intSize,hipMemcpyHostToDevice);

        // set up constants struct for copying into device memory struct
        GlobalConstants constParams;
        constParams.numBoids = numBoids;
        constParams.position = cudaDevicePosition; 
        constParams.velocity = cudaDeviceVelocity;
        constParams.flockID = cudaDeviceFlockID;
        constParams.flockSize = flockSize;

        hipMemcpyToSymbol(HIP_SYMBOL(cuConstBoidData), &constParams, sizeof(GlobalConstants));
        hipMemcpyToSymbol(HIP_SYMBOL(cuConstGlobalParams), &Params, sizeof(GlobalParams));

        auto EndTime = std::chrono::system_clock::now();
        std::chrono::duration<double> ElapsedTime = EndTime - StartTime;

        return ElapsedTime.count(); // return wall clock time diff
    }

    double Tick()
    {
        // Run our actual problem (boid computation)
        auto StartTime = std::chrono::system_clock::now();

        const int threadsPerBlock = 512;
        const int numBlocks = (numBoids + threadsPerBlock - 1) / threadsPerBlock;

        sensePlanActKernel<<<numBlocks,threadsPerBlock>>>();
        hipDeviceSynchronize();

        hipMemcpy(position, 
                   cuConstBoidData.position,
                   numBoids * sizeof(float) * 2,
                   hipMemcpyDeviceToHost);
        hipMemcpy(velocity, 
                   cuConstBoidData.velocity,
                   numBoids * sizeof(float) * 2,
                   hipMemcpyDeviceToHost);

        UpdateBoidPosAndVel();          

        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].Delegate(omp_get_thread_num(), AllFlocks);
        }

        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].AssignToFlock(omp_get_thread_num(), AllFlocks);
        }

        auto EndTime = std::chrono::system_clock::now();
        std::chrono::duration<double> ElapsedTime = EndTime - StartTime;
        // save tracer data
        Tracer::AddTickT(ElapsedTime.count());

        if (Params.RenderingMovie)
        {
            // Rendering is not part of our problem
            Render();
        }

        return ElapsedTime.count(); // return wall clock time diff
    }
    
    void Render()
    {
        // draw all the boids onto the frame
#pragma omp parallel for num_threads(Params.NumThreads) schedule(static)
        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].Draw(I);
        }
        // draw the target onto the frame
        I.ExportPPMImage();
        I.Blank();
    }
};

// declaring static variables
SimulatorParamsStruct Simulator::Params;
ImageParamsStruct Image::Params;
TracerParamsStruct Tracer::Params;


// global params struct
ParamsStruct GlobalParams;

int main()
{
    std::srand(0); // consistent seed
    ParseParams("params/params.ini");
    Simulator Sim;
    Sim.Simulate();
    // Dump all tracer data
    Tracer::Dump();
    return 0;
}