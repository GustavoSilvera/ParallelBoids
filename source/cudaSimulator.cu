#include "hip/hip_runtime.h"
#include "Flock.hpp"  // Flocks
#include "Tracer.hpp" // Tracer
#include "Utils.hpp"  // Params
#include "Vec.hpp"    // Vec3D
#include <chrono>     // timing threads
#include <omp.h>      // OpenMP
#include <string>     // cout
#include <vector>     // std::vector

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

struct GlobalBoidData {
    size_t numBoids;

    float* position;
    float* velocity;
    float* acceleration;
    int* flockID; 
    int* flockSize;
};

__constant__ GlobalBoidData cuConstBoidData;
__constant__ ParamsStruct cuConstGlobalParams;

/** @return the result of adding the inputs together */
__device__ float2 operator+(const float2 A, const float2 B)
{ 
    return make_float2(A.x + B.x, A.y + B.y);
}

/** @return the result of subtracting B from A */
__device__ float2 operator-(const float2 A, const float2 B)
{
    return make_float2(A.x - B.x, A.y - B.y);
}

/** @return the result of multiplying each element of A by v */
__device__ float2 operator*(const float2 A, const float v) 
{
    return make_float2(A.x * v, A.y * v);
}

/** @return the result of dividing each element of A by v */
__device__ float2 operator/(const float2 A, const float v) 
{
    return make_float2(A.x / v, A.y / v);
}

/** @return the square of the l2 norm of the specified vector */
__device__ float sizeSqrd (float2 A) 
{
    return A.x*A.x + A.y*A.y;
}

/** @return whether or not A lies beyond a distance of radius R from B */
__device__ bool distGT (const float2 A, const float2 B, const float R)
{
    // printf("A: (%.4f,%.4f)  || B: (%.4f,%.4f) || sizeSqrd(A-B): %.4f || R*R: %.4f\n", A.x,A.y,B.x,B.y,sizeSqrd(A-B),R*R);
    return sizeSqrd(A-B) > (R*R);
}

/** @return whether or not A lies within a distance of radius R from B */
__device__ bool distLT (const float2 A, const float2 B, const float R)
{
    return sizeSqrd(A-B) < R*R;
}

__device__ float2 normalize(const float2 A) {
    return A / (sqrt(double(sizeSqrd(A))));
}

__device__ float2 limitMagnitudeKernel(const float2 A, const double maxMag)
{
    if (sizeSqrd(A) > maxMag * maxMag)
    {
        return normalize(A) * maxMag;
    }
    return A;
}
/** 
 * Computes the acceleration for the boid indexed by index in the current
 * tick.
 */ 
__global__ void 
senseAndPlanKernel() 
{
    const int index = blockIdx.x + threadIdx.x;
    int N = cuConstBoidData.numBoids;
    float2 a1,a2,a3;
    float2* position = (float2*)cuConstBoidData.position;
    float2* velocity = (float2*)cuConstBoidData.velocity;
    float2* acceleration = (float2*)cuConstBoidData.acceleration;

    //printf("%f\n",position[0].x);
    BoidParamsStruct boidParams = cuConstGlobalParams.BoidParams;

    float2 posUs = position[index];
    float2 velUs = velocity[index];

    float2 relCOM = make_float2(0.0,0.0);  
    float2 relCOV = make_float2(0.0,0.0);
    float2 sep = make_float2(0.0,0.0);

    // printf("nborradius: %f\n", boidParams.NeighbourhoodRadius);
    int numCloseBy = 0;
    for(int i = 0; i < N; i++) {
        float2 posThem = position[i];
        float2 velThem = velocity[i];
        if (i != index 
            && !distGT(posUs,posThem, boidParams.NeighbourhoodRadius)) 
        {
            // printf("N: %d || i: %d || index: %d || posUs: %8.4f || posThem: %8.4f || relCOM: %8.4f || sep: %8.4f || relCOV: %8.4f\n",
            //        N, i, index, posUs.x, posThem.x, relCOM.x,sep.x,relCOV.x);
            relCOM = relCOM + posThem;  
            relCOV = relCOV + velThem;
            if (distLT(posUs,posThem,boidParams.CollisionRadius))
            {
                sep = sep - (posThem - posUs);
            }
            numCloseBy++;
        }
        // else printf("%d Us: %d || Them: %d\n",N, index, i);
        // else printf("Us: %d || Them: %d || GT?: %d\n",index,i,(int)(distGT(posUs,posThem,boidParams.NeighbourhoodRadius)));//printf("posUS: (%f,%f) || posThem: (%f,%f)\n", posUs.x,posUs.y, posThem.x,posUs.y);
    }

    if (numCloseBy > 0)
    {
        // printf("relCOM: %f || sep: %f || relCOV: %f\n", relCOM.x,sep.x,relCOV.x);
        a1 = ((relCOM / numCloseBy) - posUs) * (boidParams.Cohesion);
        // printf("%f\n", (((relCOM / numCloseBy) - posUs)* (boidParams.Cohesion)).x);
        a2 = sep * (boidParams.Separation); // dosent depent on NumCloseby but makes sense
        a3 = ((relCOV / numCloseBy) - velUs) * (boidParams.Alignment);
    }
    
    // printf("Vel ptr: %p\n Acc val : ",((float*)cuConstBoidData.velocity));
    // // printf("Vel val: %f\n",*((float*)cuConstBoidData.velocity));
    // printf("Acc ptr: %p\n Acc val : ",((float*)cuConstBoidData.acceleration));
    // printf("Acc val: %f\n",((float*)acceleration)[0]);
    // update the acceleration of the boid
    // float ab4 = acceleration[index].x;
    // printf("a1: %f || a2: %f || a3: %f\n", a1.x,a2.x,a3.x);
    acceleration[index] = a1 + a2 + a3;
    // printf("Before: %f ||| After: %f\n",ab4, ((float*)acceleration)[0]);
}

__global__ void 
actKernel(double deltaTime)
{
    const int index = blockIdx.x + threadIdx.x;
    /// NOTE: This function is meant to be independent from all other boids
    /// and thus can be run asynchronously, however it needs a barrier between itself
    /// and the senseAndPlan() device function
    float2* position = (float2*)cuConstBoidData.position;
    float2* velocity = (float2*)cuConstBoidData.velocity;
    float2* acceleration = (float2*)cuConstBoidData.acceleration;

    BoidParamsStruct params = cuConstGlobalParams.BoidParams;

    velocity[index] = limitMagnitudeKernel((velocity[index] + acceleration[index]), params.MaxVel);
    // float2 before = position[index];
    position[index] = position[index] + (velocity[index] * deltaTime);
    // printf("Before: (%.4f,%.4f) || After: (%.4f,%.4f)\n", before.x,before.y,position[index].x,position[index].y);
}

class Simulator
{
  public:
    GlobalBoidData cuDevicePtrs;
    float* cudaDevicePosition;
    float* cudaDeviceVelocity;
    float* cudaDeviceAcceleration;
    int* cudaDeviceFlockID;
    int* cudaDeviceFlockSize;

    int numBoids;
    float* position;
    float* velocity;
    float* acceleration;
    int* flockID;
    int* flockSize;
    Simulator()
    {
        Params = GlobalParams.SimulatorParams;
        // Print out status
        std::cout << "Running on " << Params.NumBoids << " boids for " << Params.NumIterations << " iterations in a ("
                  << GlobalParams.ImageParams.WindowX << ", " << GlobalParams.ImageParams.WindowY << ") world with "
                  << Params.NumThreads << " threads" << std::endl;

        // Initialize neighbourhood layout for flocks before use
        Flock::InitNeighbourhoodLayout();
        // Spawn flocks
        for (size_t i = 0; i < Params.NumBoids; i++)
        {
            AllFlocks.push_back(Flock(i, 1));
        }

        // begin tracking which flocks communicate with which
        Tracer::InitFlockMatrix(AllFlocks.size());

        // Allocate and initialize device memory data
        Setup();

        // initialize image frame
        if (Params.RenderingMovie)
        {
            // only allocate memory if we're gonna use it
            I.Init();
        }
    }
    static SimulatorParamsStruct Params;
    std::vector<Flock> AllFlocks;
    std::vector<int> FlockSizes;
    Image I;

    void Simulate()
    {
        double ElapsedTime = 0;
        for (size_t i = 0; i < Params.NumIterations; i++)
        {
            ElapsedTime += Tick();
            std::cout << "Tick: " << i << "\r" << std::flush; // carriage return, no newline
        }
        std::cout << "Finished simulation! Took " << ElapsedTime << "s" << std::endl;
    }

    /**
     * Inserts the value v in the float2 array represented by arr
     * @pre i < |arr|/2
     */
    void insertFloat2(float* arr, Vec2D v, size_t i) {
        arr[2*i] = v[0];
        arr[2*i+1] = v[1];
    }

    void InitBoidDataArrays() {
        std::vector<Boid> &AllBoids = *(AllFlocks.begin()->Neighbourhood.GetAllBoidsPtr());
        size_t vecSize = sizeof(float) * 2 * numBoids;
        size_t intSize = sizeof(int) * numBoids;

        position = (float*)malloc(vecSize);
        velocity = (float*)malloc(vecSize);
        acceleration = (float*)malloc(vecSize);
        flockID = (int*)malloc(intSize);
        flockSize = (int*)malloc(intSize);

        for (size_t i=0; i < AllBoids.size(); i++)
        {
            Boid &boid = AllBoids[i];
            Vec2D pos = boid.Position;
            Vec2D vel = boid.Velocity;
            Vec2D acc = boid.Acceleration;
            int fID = boid.FlockID;
            int fSize = 1;

            insertFloat2(position, pos, i);
            insertFloat2(velocity, vel, i);
            insertFloat2(acceleration, acc,i);
            flockID[i] = fID;
            flockSize[i] = fSize;
        }
    }

    /**
     * @pre All Boid arrays have been initialized and are up to date.
     */
    void UpdateBoidPosAndVel() {
        std::vector<Boid> &AllBoids = *(AllFlocks.begin()->Neighbourhood.GetAllBoidsPtr());
        for (size_t i = 0; i < AllBoids.size(); i++)
        {
            Boid &B = AllBoids[i];
            B.Position = Vec2D(position[2*i],position[2*i+1]);
            B.Velocity = Vec2D(velocity[2*i],velocity[2*i+1]);
        }
    }

    /** 
     * Sets up the memory required for the cuda device code
     *
     * @return The time it took to setup. 
     */
    double Setup() {
        auto StartTime = std::chrono::system_clock::now();

        // allocate boid and flock memory
        std::vector<Boid> &AllBoids = *(AllFlocks.begin()->Neighbourhood.GetAllBoidsPtr());
        numBoids = AllBoids.size();
        size_t vecSize = sizeof(float) * 2 * numBoids;
        size_t intSize = sizeof(int) * numBoids;
        hipMalloc(&cudaDevicePosition, vecSize);
        hipMalloc(&cudaDeviceVelocity, vecSize);
        hipMalloc(&cudaDeviceAcceleration, vecSize);
        hipMalloc(&cudaDeviceFlockID, intSize);
        hipMalloc(&flockSize, intSize);

        InitBoidDataArrays();
        
        hipMemcpy(cudaDevicePosition,position,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceVelocity,velocity,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceAcceleration,acceleration,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceFlockID,flockID,intSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceFlockSize,flockSize,intSize,hipMemcpyHostToDevice);

        // set up constants struct for copying into device memory struct
        cuDevicePtrs.numBoids = numBoids;
        cuDevicePtrs.position = cudaDevicePosition; 
        cuDevicePtrs.velocity = cudaDeviceVelocity;
        cuDevicePtrs.acceleration = cudaDeviceAcceleration;
        cuDevicePtrs.flockID = cudaDeviceFlockID;
        cuDevicePtrs.flockSize = flockSize;

        hipMemcpyToSymbol(HIP_SYMBOL(cuConstBoidData), &cuDevicePtrs, sizeof(GlobalBoidData));
        hipMemcpyToSymbol(HIP_SYMBOL(cuConstGlobalParams), &GlobalParams, sizeof(ParamsStruct));

        auto EndTime = std::chrono::system_clock::now();
        std::chrono::duration<double> ElapsedTime = EndTime - StartTime;

        return ElapsedTime.count(); // return wall clock time diff
    }

    double Tick()
    {
        // Run our actual problem (boid computation)
        auto StartTime = std::chrono::system_clock::now();

        const int threadsPerBlock = 512;
        const int numBlocks = (numBoids + threadsPerBlock - 1) / threadsPerBlock;

        senseAndPlanKernel<<<numBlocks,threadsPerBlock>>>();
        hipDeviceSynchronize();
        actKernel<<<numBlocks,threadsPerBlock>>>(Params.DeltaTime);
        hipDeviceSynchronize();

        // GlobalBoidData Results;
        // hipMemcpyFromSymbol(&Results.position, 
        HIP_SYMBOL(//            "cuConstBoidData.position"),
        //            sizeof(float*),
        //            0, hipMemcpyDeviceToHost);
        // printf("%p", Results.position);
        hipMemcpy(position, 
                   cuDevicePtrs.position,
                   numBoids * sizeof(float) * 2,
                   hipMemcpyDeviceToHost);
        hipMemcpy(velocity, 
                   cuDevicePtrs.velocity,
                   numBoids * sizeof(float) * 2,
                   hipMemcpyDeviceToHost);

        //printf("%f\n",position[0]);
        UpdateBoidPosAndVel();     
#pragma omp parallel num_threads(Params.NumThreads)
    {
#pragma for schedule(static)
        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].Delegate(omp_get_thread_num(), AllFlocks);
        }
#pragma omp barrier
#pragma for schedule(static)
        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].AssignToFlock(omp_get_thread_num(), AllFlocks);
        }
    }
        auto EndTime = std::chrono::system_clock::now();
        std::chrono::duration<double> ElapsedTime = EndTime - StartTime;
        // save tracer data
        Tracer::AddTickT(ElapsedTime.count());

        if (Params.RenderingMovie)
        {
            // Rendering is not part of our problem
            Render();
        }

        return ElapsedTime.count(); // return wall clock time diff
    }
    
    void Render()
    {
        // draw all the boids onto the frame
#pragma omp parallel for num_threads(Params.NumThreads) schedule(static)
        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].Draw(I);
        }
        // draw the target onto the frame
        I.ExportPPMImage();
        I.Blank();
    }
};

// declaring static variables
SimulatorParamsStruct Simulator::Params;
ImageParamsStruct Image::Params;
TracerParamsStruct Tracer::Params;


// global params struct
ParamsStruct GlobalParams;

int main()
{
    std::srand(0); // consistent seed
    ParseParams("params/params.ini");
    Simulator Sim;
    Sim.Simulate();
    // Dump all tracer data
    Tracer::Dump();
    return 0;
}