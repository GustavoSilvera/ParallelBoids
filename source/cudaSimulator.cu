#include "hip/hip_runtime.h"
#include "Flock.hpp"  // Flocks
#include "Tracer.hpp" // Tracer
#include "Utils.hpp"  // Params
#include "Vec.hpp"    // Vec3D
#include <chrono>     // timing threads
#include <omp.h>      // OpenMP
#include <string>     // cout
#include <vector>     // std::vector

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

struct GlobalConstants {
    size_t numBoids;

    float* position;
    float* velocity;
    int* flockID; 
    int* flockSize;
};

__constant__ GlobalConstants cuConstTickParams;
class Simulator
{
  public:
    float* cudaDevicePosition;
    float* cudaDeviceVelocity;
    int* cudaDeviceFlockID;
    int* cudaDeviceFlockSize;

    float* position;
    float* velocity;
    int* flockID;
    int* flockSize;
    Simulator()
    {
        Params = GlobalParams.SimulatorParams;
        // Print out status
        std::cout << "Running on " << Params.NumBoids << " boids for " << Params.NumIterations << " iterations in a ("
                  << GlobalParams.ImageParams.WindowX << ", " << GlobalParams.ImageParams.WindowY << ") world with "
                  << Params.NumThreads << " threads" << std::endl;

        // Initialize neighbourhood layout for flocks before use
        Flock::InitNeighbourhoodLayout();
        // Spawn flocks
        for (size_t i = 0; i < Params.NumBoids; i++)
        {
            AllFlocks.push_back(Flock(i, 1));
        }

        // begin tracking which flocks communicate with which
        Tracer::InitFlockMatrix(AllFlocks.size());

        // initialize image frame
        if (Params.RenderingMovie)
        {
            // only allocate memory if we're gonna use it
            I.Init();
        }
    }
    static SimulatorParamsStruct Params;
    std::vector<Flock> AllFlocks;
    std::vector<int> FlockSizes;
    Image I;

    void Simulate()
    {
        double ElapsedTime = 0;
        for (size_t i = 0; i < Params.NumIterations; i++)
        {
            ElapsedTime += Tick();
            std::cout << "Tick: " << i << "\r" << std::flush; // carriage return, no newline
        }
        std::cout << "Finished simulation! Took " << ElapsedTime << "s" << std::endl;
    }

    /**
     * Inserts the value v in the float2 array represented by arr
     * @pre i < |arr|/2
     */
    void insertFloat2(float* arr, Vec2D v, size_t i) {
        arr[2*i] = v[0];
        arr[2*i+1] = v[1];
    }

    void InitBoidDataArrays() {
        std::vector<Boid> &AllBoids = *(AllFlocks.begin()->Neighbourhood.GetAllBoidsPtr());
        size_t numBoids = AllBoids.size();
        size_t vecSize = sizeof(float) * 2 * numBoids;
        size_t intSize = sizeof(int) * numBoids;

        position = (float*)malloc(vecSize);
        velocity = (float*)malloc(vecSize);
        flockID = (int*)malloc(intSize);
        flockSize = (int*)malloc(intSize);

        for (size_t i=0; i < AllBoids.size(); i++)
        {
            Boid boid = AllBoids[i];
            Vec2D pos = boid.Position;
            Vec2D vel = boid.Velocity;
            int fID = boid.FlockID;
            int fSize = 1;

            insertFloat2(position, pos, i);
            insertFloat2(velocity, vel, i);
            flockID[i] = fID;
            flockSize[i] = fSize;
        }
    }
    /** 
     * Sets up the memory required for the cuda device code
     *
     * @return The time it took to setup. 
     */
    double Setup() {
        auto StartTime = std::chrono::system_clock::now();

        // allocate boid and flock memory
        std::vector<Boid> &AllBoids = *(AllFlocks.begin()->Neighbourhood.GetAllBoidsPtr());
        size_t numBoids = AllBoids.size();
        size_t vecSize = sizeof(float) * 2 * numBoids;
        size_t intSize = sizeof(int) * numBoids;
        hipMalloc(&cudaDevicePosition, vecSize);
        hipMalloc(&cudaDeviceVelocity, vecSize);
        hipMalloc(&cudaDeviceFlockID, intSize);
        hipMalloc(&flockSize, intSize);

        InitBoidDataArrays();
        
        hipMemcpy(cudaDevicePosition,position,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceVelocity,velocity,vecSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceFlockID,flockID,intSize,hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceFlockSize,flockSize,intSize,hipMemcpyHostToDevice);

        // set up constants struct for copying into device memory struct
        GlobalConstants constParams;
        constParams.numBoids = numBoids;
        constParams.position = cudaDevicePosition; 
        constParams.velocity = cudaDeviceVelocity;
        constParams.flockID = cudaDeviceFlockID;
        constParams.flockSize = flockSize;

        hipMemcpyToSymbol(HIP_SYMBOL(cuConstTickParams), &constParams, sizeof(GlobalConstants));

        auto EndTime = std::chrono::system_clock::now();
        std::chrono::duration<double> ElapsedTime = EndTime - StartTime;

        return ElapsedTime.count(); // return wall clock time diff
    }

    double Tick()
    {
        // Run our actual problem (boid computation)
        auto StartTime = std::chrono::system_clock::now();
        auto EndTime = std::chrono::system_clock::now();
        std::chrono::duration<double> ElapsedTime = EndTime - StartTime;
        // save tracer data
        Tracer::AddTickT(ElapsedTime.count());

        if (Params.RenderingMovie)
        {
            // Rendering is not part of our problem
            Render();
        }

        return ElapsedTime.count(); // return wall clock time diff
    }
    
    void Render()
    {
        // draw all the boids onto the frame
#pragma omp parallel for num_threads(Params.NumThreads) schedule(static)
        for (size_t i = 0; i < AllFlocks.size(); i++)
        {
            AllFlocks[i].Draw(I);
        }
        // draw the target onto the frame
        I.ExportPPMImage();
        I.Blank();
    }
};

// declaring static variables
SimulatorParamsStruct Simulator::Params;
ImageParamsStruct Image::Params;
TracerParamsStruct Tracer::Params;


// global params struct
ParamsStruct GlobalParams;

int main()
{
    std::srand(0); // consistent seed
    ParseParams("params/params.ini");
    Simulator Sim;
    Sim.Simulate();
    // Dump all tracer data
    Tracer::Dump();
    return 0;
}